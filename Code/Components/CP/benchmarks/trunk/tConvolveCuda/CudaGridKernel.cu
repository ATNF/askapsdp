#include "hip/hip_runtime.h"
// @copyright (c) 2009 CSIRO
// Australia Telescope National Facility (ATNF)
// Commonwealth Scientific and Industrial Research Organisation (CSIRO)
// PO Box 76, Epping NSW 1710, Australia
// atnf-enquiries@csiro.au
//
// This file is part of the ASKAP software distribution.
//
// The ASKAP software distribution is free software: you can redistribute it
// and/or modify it under the terms of the GNU General Public License as
// published by the Free Software Foundation; either version 2 of the License,
// or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program; if not, write to the Free Software
// Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
//
// @author Ben Humphreys <ben.humphreys@csiro.au>
// @author Tim Cornwell  <tim.cornwell@csiro.au>

// System includes
#include <stdio.h>

// Local includes
#include "CudaGridKernel.h"

// Constants
static const int cg_maxSupport = 256;

// Check and report last error
__host__ __inline__ void checkError(void)
{
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
                printf("CUDA Error: %s\n", hipGetErrorString(err));
        }
}

// Perform Gridding (Device Function)
// Each thread handles a different grid point
__global__ void d_gridKernel(const Complex *data, const int support,
		const Complex *C, const int *cOffset,
		const int *iu, const int *iv,
		Complex *grid, const int gSize, const int dind)
{
	// The actual starting grid point
	__shared__ int s_gind;
	// The Convoluton function point from which we offset
	__shared__ int s_cind;

	// Calculate the data index offset for this block
	const int l_dind = dind + blockIdx.y;

	// A copy of the vis data so all threads can read it from shared
	// memory rather than all reading from device memory.
	__shared__ Complex l_data;

	if (threadIdx.x == 0) {
		s_gind = iu[l_dind] + gSize * iv[l_dind] - support;
		s_cind = cOffset[l_dind];
		l_data = data[l_dind];
	}
	__syncthreads();

	// Make a local copy from shared memory
	int gind = s_gind;
	int cind = s_cind;

	// blockIdx.x gives the support location in the v direction
	int sSize = 2 * support + 1;
	gind += gSize * blockIdx.x;
	cind += sSize * blockIdx.x;

	// threadIdx.x gives the support location in the u dirction
	grid[gind+threadIdx.x] = hipCaddf(grid[gind+threadIdx.x], hipCmulf(l_data, C[cind+threadIdx.x]));
}

// Perform Gridding (Host Function)
__host__ void cuda_gridKernel(const Complex  *data, const int dSize, const int support,
		const Complex *C, const int *cOffset,
		const int *iu, const int *iv,
		Complex *grid, const int gSize,
		const int *h_iu, const int *h_iv)
{
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(d_gridKernel), hipFuncCachePreferL1);

	const int sSize=2*support+1;
	int step = 1;

	// This loop begs some explanation. It steps through each spectral
	// sample either one at a time or two at a time. It will do two samples
	// if the two regions involved do not overlap. If they do, only a 
	// single point is gridded.
	//
	// Gridding two point is better than one because giving the GPU more
	// work to do allows it to hide memory latency better.
	for (int dind = 0; dind < dSize; dind += step) {
		if ((dind+1) < dSize && (
		(h_iu[dind] - h_iu[dind+1]) > sSize ||
		(h_iv[dind] - h_iv[dind+1]) > sSize)) {
			step = 2;
        		dim3 gridDim(sSize, step);
			d_gridKernel<<< gridDim, sSize >>>(data, support,
					C, cOffset, iu, iv, grid, gSize, dind);
                	checkError();
		} else {
			step = 1;
        		dim3 gridDim(sSize, step);
			d_gridKernel<<< gridDim, sSize >>>(data, support,
				C, cOffset, iu, iv, grid, gSize, dind);
                	checkError();
		}
	}
}

// Perform De-Gridding (Device Function)
__global__ void d_degridKernel(const Complex *grid, const int gSize, const int support,
                const Complex *C, const int *cOffset,
                const int *iu, const int *iv,
                Complex  *data, const int dind,
		int row)
{
	// Private data for each thread. Eventually summed by the
	// master thread (i.e. threadIdx.x == 0). Currently 
	__shared__ Complex s_data[cg_maxSupport];
	s_data[threadIdx.x] = make_hipFloatComplex(0, 0);

	const int l_dind = dind + blockIdx.x;

        // The actual starting grid point
        __shared__ int s_gind;
        // The Convoluton function point from which we offset
        __shared__ int s_cind;

        if (threadIdx.x == 0) {
                s_gind = iu[l_dind] + gSize * iv[l_dind] - support;
                s_cind = cOffset[l_dind];
        }
        __syncthreads();

        // Make a local copy from shared memory
        int gind = s_gind;
        int cind = s_cind;

        // row gives the support location in the v direction
        int sSize = 2 * support + 1;
        gind += gSize * row;
        cind += sSize * row;

	// threadIdx.x gives the support location in the u dirction
	s_data[threadIdx.x] = hipCmulf(grid[gind+threadIdx.x], C[cind+threadIdx.x]);

	// Sum all the private data elements and accumulate to the
	// device memory
        __syncthreads();
	if (threadIdx.x == 0) {
		Complex sum = make_hipFloatComplex(0, 0);
		Complex original;
		original = data[l_dind];
		#pragma unroll 129
		for (int i = 0; i < sSize; ++i) {
			sum = hipCaddf(sum, s_data[i]);
		}
		original = hipCaddf(original, sum);
		data[l_dind] = original;
	}
}

// Perform De-Gridding (Host Function)
__host__ void cuda_degridKernel(const Complex *grid, const int gSize, const int support,
                const Complex *C, const int *cOffset,
                const int *iu, const int *iv,
                Complex  *data, const int dSize)
{
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(d_degridKernel), hipFuncCachePreferL1);

    int sSize = 2 * support + 1;
	if (sSize > cg_maxSupport) {
		printf("Support size of %d exceeds max support size of %d\n",
			sSize, cg_maxSupport);
	}

	int dimGrid = 4096;	// 4096 is starting size
	for (int dind = 0; dind < dSize; dind += dimGrid) {
		if ((dSize - dind) < dimGrid) {
            // If there are less than 4096 elements left,
            // just do the remaining
            dimGrid = dSize - dind;
        }

        for (int row = 0; row < sSize; ++row) {
            d_degridKernel<<< dimGrid, sSize >>>(grid, gSize, support,
                    C, cOffset, iu, iv, data, dind, row);
            checkError();
        }
    }
}
